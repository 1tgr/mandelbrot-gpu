#include "hip/hip_runtime.h"
#include <stdio.h>
#include <vector>

#include "thrust/device_vector.h"
#include "thrust/host_vector.h"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb/stb_image_write.h"

#include "../mandel.h"
#include "../timer.h"

static __global__ void mandel_kernel(char *image, int width, int height) {
    auto x = blockIdx.x * blockDim.x + threadIdx.x;
    auto y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height) {
        mandel(image, width, height, x, y);
    }
}

int main() {
    auto timer = Timer();
    const auto width = 8400;
    const auto height = 4800;
    auto image = std::vector<char>(width * height * 3);
    dim3 blockDim(16, 16, 1);
    dim3 gridDim((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y, 1);
    auto device_image = thrust::device_vector<char>(image.size());
    printf("[%f] Init\n", timer.measure());
    mandel_kernel<<<gridDim, blockDim, 0>>>(thrust::raw_pointer_cast(&device_image[0]), width, height);
    thrust::copy(device_image.begin(), device_image.end(), image.begin());
    printf("[%f] Kernel (GPU)\n", timer.measure());
    stbi_write_png("image.png", width, height, 3, &image[0], width * 3);
    printf("[%f] .png\n", timer.measure());
    return 0;
}
